#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/**
 * Copyright (C) Mellanox Technologies Ltd. 2021.  ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include "../mc_rocm.h"
#ifdef __cplusplus
}
#endif

__global__ void wait_kernel(volatile uint32_t *status) {
    ucc_status_t st;
    *status = UCC_MC_ROCM_TASK_STARTED;
    do {
        st = (ucc_status_t)*status;
    } while(st != UCC_MC_ROCM_TASK_COMPLETED);
    *status = UCC_MC_ROCM_TASK_COMPLETED_ACK;
    return;
}

__global__ void wait_kernel_nb(volatile uint32_t *status) {
    *status = UCC_MC_ROCM_TASK_COMPLETED_ACK;
    return;
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t ucc_mc_rocm_post_kernel_stream_task(uint32_t *status,
                                                 int blocking_wait,
                                                 hipStream_t stream)
{
    if (blocking_wait) {
        hipLaunchKernelGGL(wait_kernel, dim3(1), dim3(1), 0, stream, status);
    } else {
        hipLaunchKernelGGL(wait_kernel_nb, dim3(1), dim3(1), 0, stream, status);
    }
    ROCMCHECK(hipGetLastError());
    return UCC_OK;
}

#ifdef __cplusplus
}
#endif
